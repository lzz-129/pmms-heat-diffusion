#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "timer.h"

#define image_height 1024
#define image_width 1024
#define filter_height 5
#define filter_width 5

#define border_height ((filter_height/2)*2)
#define border_width ((filter_width/2)*2)
#define input_height (image_height + border_height)
#define input_width (image_width + border_width)

#define block_size_x 32
#define block_size_y 16

#define SEED 1234

using namespace std;

void convolutionSeq(float *output, float *input, float *filter) {
    //for each pixel in the output image

  timer sequentialTime = timer("Sequential");
  
  sequentialTime.start();

    for (int y=0; y < image_height; y++) {
        for (int x=0; x < image_width; x++) { 
	    output[y*image_width+x]=0;
            //for each filter weight
            for (int i=0; i < filter_height; i++) {
                for (int j=0; j < filter_width; j++) {
                    output[y*image_width+x] += input[(y+i)*input_width+x+j] * filter[i*filter_width+j];
                }
            }
	    output[y*image_width+x] /= 35;
        }
    }
  
  sequentialTime.stop(); 
  cout << "convolution (sequential): \t\t" << sequentialTime << endl;

}


__global__ void convolution_kernel_naive(float *output, float *input, float *filter) {

}

void convolutionCUDA(float *output, float *input, float *filter) {
    float *d_input; float *d_output; float *d_filter;
    hipError_t err;
    timer kernelTime = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // memory allocation
    err = hipMalloc((void **)&d_input, input_height*input_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_input: %s\n", hipGetErrorString( err )); }
    err = hipMalloc((void **)&d_output, image_height*image_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString( err )); }
    err = hipMalloc((void **)&d_filter, filter_height*filter_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_filter: %s\n", hipGetErrorString( err )); }

    memoryTime.start();
    // host to device 
    err = hipMemcpy(d_input, input, input_height*input_width*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy host to device input: %s\n", hipGetErrorString( err ));  }
    err = hipMemcpy(d_filter, filter, filter_height*filter_width*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy host to device filter: %s\n", hipGetErrorString( err ));  }
    
    // zero the result array 
    err = hipMemset(d_output, 0, image_height*image_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemset output: %s\n", hipGetErrorString( err ));  }
    memoryTime.stop();
    //setup the grid and thread blocks
    //thread block size
    dim3 threads(block_size_x, block_size_y);
    //problem size divided by thread block size rounded up
    dim3 grid(int(ceilf(image_width/(float)threads.x)), int(ceilf(image_height/(float)threads.y)) );

    //measure the GPU function
    kernelTime.start();
    convolution_kernel_naive<<<grid, threads>>>(d_output, d_input, d_filter);
    hipDeviceSynchronize();
    kernelTime.stop();
 
    //check to see if all went well
    err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "Error during kernel launch convolution_kernel: %s\n", hipGetErrorString( err )); }

    //copy the result back to host memory
    memoryTime.start();
    err = hipMemcpy(output, d_output, image_height*image_width*sizeof(float), hipMemcpyDeviceToHost);
    memoryTime.stop();
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy device to host output: %s\n", hipGetErrorString( err )); }
 
    err = hipFree(d_input);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_input: %s\n", hipGetErrorString( err )); }
    err = hipFree(d_output);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString( err )); }
    err = hipFree(d_filter);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_filter: %s\n", hipGetErrorString( err )); }

    cout << "convolution (kernel): \t\t" << kernelTime << endl;
    cout << "convolution (memory): \t\t" << memoryTime << endl;

}

int compare_arrays(float *a1, float *a2, int n) {
    int errors = 0;
    int print = 0;

    for (int i=0; i<n; i++) {

        if (isnan(a1[i]) || isnan(a2[i])) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n",i,a1[i],a2[i]);
            }
        }

        float diff = (a1[i]-a2[i])/a1[i];
        if (diff > 1e-6f) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t rel_error=\t %10.7e\n",i,a1[i],a2[i],diff);
            }
        }

    }

    return errors;
}
        

int main() {
    int i; 
    int errors=0;

    //allocate arrays and fill them
    float *input = (float *) malloc(input_height * input_width * sizeof(float));
    float *output1 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output2 = (float *) calloc(image_height * image_width, sizeof(float));
    float *filter = (float *) malloc(filter_height * filter_width * sizeof(float));

    for (i=0; i< input_height * input_width; i++) {
        input[i] = (float) (i % SEED);
    }

//THis is specific for a W==H smoothening filteri, where W and H are odd.
    for (i=0; i<filter_height * filter_width; i++) { 
      filter[i] = 1.0;
    }

    for (i=filter_width+1; i<(filter_height - 1) * filter_width; i++) {
	if (i % filter_width > 0 && i % filter_width < filter_width-1) filter[i]+=1.0; 
    }

    filter[filter_width*filter_height/2]=3.0;
//end initialization
   
    //measure the CPU function
    convolutionSeq(output1, input, filter);
    //measure the GPU function
    convolutionCUDA(output2, input, filter);


    //check the result
    errors += compare_arrays(output1, output2, image_height*image_width);
    if (errors > 0) {
        printf("TEST FAILED! %d errors!\n", errors);
    } else {
        printf("TEST PASSED!\n");
    }


    free(filter);
    free(input);
    free(output1);
    free(output2);

    return 0;
}


