#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include "timer.h"

enum {
    image_height = 1024,
    image_width = 1024,
    filter_height = 5,
    filter_width = 5,
    border_height =  filter_height & -2,
    border_width = filter_width & -2,
    input_height = image_height + border_height,
    input_width = image_width + border_width,
    block_size_x = 32,
    block_size_y = 8,
    sh_input_height = block_size_y + border_height,
    sh_input_width = block_size_x + border_width,
    SEED = 1234
};

using std::isnan;
using std::fprintf;
using std::printf;
using std::puts;
using std::cout;
using std::endl;

static void convolutionSeq(float *output, float *input, float *filter);
__global__ void convolution_kernel_naive(float *output, float *input, float *filter);
static void convolutionCUDA(float *output, float *input, float *filter);
static int compare_arrays(float *a1, float *a2, int n);

int main()
{
    // Allocate arrays and fill them
    float *input = new float[input_height * input_width];
    float *output1 = new float[image_height * image_width];
    float *output2 = new float[image_height * image_width];
    float *filter = new float[filter_height * filter_width];
    for (int i = 0; i < input_height * input_width; ++i) {
        input[i] = static_cast<float>(i % SEED);
    }
// This is specific for a W == H smoothening filter i, where W and H are odd.
    for (int i = 0; i < filter_height * filter_width; ++i) {
        filter[i] = 1.0f;
    }
    for (int i = filter_width + 1; i < (filter_height - 1) * filter_width; ++i) {
	    if (i % filter_width > 0 && i % filter_width < filter_width - 1)
            filter[i] += 1.0f;
    }
    filter[filter_width * filter_height >> 1] = 3.0f;
// End initialisation
    // Measure the CPU function
    convolutionSeq(output1, input, filter);
    // Measure the GPU function
    convolutionCUDA(output2, input, filter);
    // Check the result
    int errors = compare_arrays(output1, output2, image_height * image_width);
    if (errors > 0)
        printf("TEST FAILED! %d errors!\n", errors);
    else
        puts("TEST PASSED!");
    delete[] input;
    delete[] output1;
    delete[] output2;
    delete[] filter;
    return 0;
}

static void convolutionSeq(float *output, float *input, float *filter)
{
    // For each pixel in the output image
    timer sequentialTime = timer("Sequential");
    sequentialTime.start();
    for (int y = 0; y < image_height; ++y) {
        for (int x = 0; x < image_width; ++x) {
	        output[y * image_width + x] = 0.0f;
            // For each filter weight
            for (int i = 0; i < filter_height; ++i) {
                for (int j = 0; j < filter_width; ++j) {
                    output[y * image_width + x] += input[(y + i) * input_width + x + j] * filter[i * filter_width + j];
                }
            }
	        output[y * image_width + x] /= 35.0f;
        }
    }
    sequentialTime.stop();
    cout << "convolution (sequential): \t\t" << sequentialTime << endl;
}

__global__ void convolution_kernel_naive(float *output, float *input, float *filter)
{
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float sh_input[sh_input_height][sh_input_width];
    if (y < input_height && x < input_width) {
        if (threadIdx.y == blockDim.y - 1) {
            if (threadIdx.x == blockDim.x - 1) {
                for (int i = 0; i <= border_height; ++i) {
                    for (int j = 0; j <= border_width; ++j) {
                        sh_input[threadIdx.y + i][threadIdx.x + j] = input[(y + i) * input_width + x + j];
                    }
                }
            } else {
                for (int i = 0; i <= border_height; ++i) {
                    sh_input[threadIdx.y + i][threadIdx.x] = input[(y + i) * input_width + x];
                }
            }
        } else if (threadIdx.x == blockDim.x - 1) {
            for (int i = 0; i <= border_width; ++i) {
                sh_input[threadIdx.y][threadIdx.x + i] = input[y * input_width + x + i];
            }
        } else {
            sh_input[threadIdx.y][threadIdx.x] = input[y * input_width + x];
        }
    }
    __shared__ float sh_filter[filter_height][filter_width];
    if (threadIdx.y < filter_height && threadIdx.x < filter_width) {
        if (threadIdx.y == blockDim.y - 1) {
            if (threadIdx.x == blockDim.x - 1) {
                for (int i = threadIdx.y; i < filter_height; ++i) {
                    for (int j = threadIdx.x; i < filter_width; ++j) {
                        sh_filter[i][j] = filter[i * filter_width + j];
                    }
                }
            } else {
                for (int i = threadIdx.y; i < filter_height; ++i) {
                    sh_filter[i][threadIdx.x] = filter[i * filter_width + threadIdx.x];
                }
            }
        } else if (threadIdx.x == blockDim.x - 1) {
            for (int i = threadIdx.x; i < filter_width; ++i) {
                sh_filter[threadIdx.y][i] = filter[threadIdx.y * filter_width + i];
            }
        } else {
            sh_filter[threadIdx.y][threadIdx.x] = filter[threadIdx.y * filter_width + threadIdx.x];
        }
    }
    __syncthreads();
    if (y < image_height && x < image_width) {
        float result = 0.0f;
        for (int i = 0; i < filter_height; ++i) {
            for (int j = 0; j < filter_width; ++j) {
                result += sh_input[threadIdx.y + i][threadIdx.x + j] * sh_filter[i][j];
            }
        }
        output[y * image_width + x] = result / 35.0f;
    }
}

static void convolutionCUDA(float *output, float *input, float *filter)
{
    // Memory allocation
    float *d_input = nullptr;
    hipError_t err = hipMalloc(&d_input, input_height * input_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_input: %s\n", hipGetErrorString(err));
    float *d_output = nullptr;
    err = hipMalloc(&d_output, image_height * image_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString(err));
    float *d_filter = nullptr;
    err = hipMalloc(&d_filter, filter_height * filter_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_filter: %s\n", hipGetErrorString(err));
    hipStream_t stream[3];
    for (int i = 0; i < 3; ++i) {
        hipStreamCreate(&stream[i]);
    }
    timer memoryTime = timer("memoryTime");
    memoryTime.start();
    // Host to device
    hipMemcpyAsync(d_input, input, input_height * input_width * sizeof(float), hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_filter, filter, filter_height * filter_width * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    // Zero the result array
    hipMemsetAsync(d_output, 0, image_height * image_width * sizeof(float), stream[2]);
    for (int i = 0; i < 3; ++i) {
        hipStreamSynchronize(stream[i]);
    }
    err = hipGetLastError();
    memoryTime.stop();
    for (int i = 0; i < 3; ++i) {
        hipStreamDestroy(stream[i]);
    }
    // Set up the grid and thread blocks
    // Thread block size
    dim3 threads(block_size_x, block_size_y);
    // Problem size divided by thread block size rounded up
    dim3 grid(static_cast<unsigned>(ceilf(image_width / static_cast<float>(threads.x))), static_cast<unsigned>(ceilf(image_height / static_cast<float>(threads.y))));
    // Measure the GPU function
    timer kernelTime = timer("kernelTime");
    kernelTime.start();
    convolution_kernel_naive<<<grid, threads>>>(d_output, d_input, d_filter);
    hipDeviceSynchronize();
    kernelTime.stop();
    // Check to see if all went well
    err = hipGetLastError();
    if (err != hipSuccess)
        fprintf(stderr, "Error during kernel launch convolution_kernel: %s\n", hipGetErrorString(err));
    // Copy the result back to host memory
    memoryTime.start();
    err = hipMemcpy(output, d_output, image_height * image_width * sizeof(float), hipMemcpyDeviceToHost);
    memoryTime.stop();
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemcpy device to host output: %s\n", hipGetErrorString(err));
    err = hipFree(d_input);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_input: %s\n", hipGetErrorString(err));
    err = hipFree(d_output);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString(err));
    err = hipFree(d_filter);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_filter: %s\n", hipGetErrorString(err));
    cout << "convolution (kernel): \t\t" << kernelTime << endl;
    cout << "convolution (memory): \t\t" << memoryTime << endl;
}

static int compare_arrays(float *a1, float *a2, int n)
{
    int errors = 0;
    int print = 0;
    for (int i = 0; i < n; ++i) {
        if (isnan(a1[i]) || isnan(a2[i])) {
            ++errors;
            if (print < 10) {
                ++print;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n", i, a1[i], a2[i]);
            }
        }
        float diff = (a1[i] - a2[i]) / a1[i];
        if (diff > 1e-6f) {
            ++errors;
            if (print < 10) {
                ++print;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t rel_error=\t %10.7e\n", i, a1[i], a2[i], diff);
            }
        }
    }
    return errors;
}
