#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
// insert operation here

}

void histogramCuda(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    int threadBlockSize = 512;
    float kernelTime = 0;
    float h2dTime, d2hTime, memTime = 0;
    hipEvent_t s1,s2,s3,s4,s5,s6;

    // allocate the vectors on the GPU
    unsigned char* deviceImage = NULL;
    checkCudaCall(hipMalloc((void **) &deviceImage, img_size * sizeof(unsigned char)));
    if (deviceImage == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    unsigned int* deviceHisto = NULL;
    checkCudaCall(hipMalloc((void **) &deviceHisto, hist_size * sizeof(unsigned int)));
    if (deviceHisto == NULL) {
        checkCudaCall(hipFree(deviceImage));
        cout << "could not allocate memory!" << endl;
        return;
    }
    hipEventCreate(&s1);
    hipEventCreate(&s2);
    hipEventCreate(&s3);
    hipEventCreate(&s4);
    hipEventCreate(&s5);
    hipEventCreate(&s6);

    // copy the original vectors to the GPU
    hipEventRecord(s1,0);
    checkCudaCall(hipMemcpy(deviceImage, image, img_size*sizeof(unsigned char), hipMemcpyHostToDevice));
    hipEventRecord(s2,0);
    
    // execute kernel
    hipEventRecord(s3,0);
    histogramKernel<<<img_size/threadBlockSize, threadBlockSize>>>(deviceImage, img_size, deviceHisto, hist_size);
    hipEventRecord(s4,0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    hipEventRecord(s5,0);
    checkCudaCall(hipMemcpy(histogram, deviceHisto, hist_size * sizeof(unsigned int), hipMemcpyDeviceToHost));
    hipEventRecord(s6,0);

    checkCudaCall(hipFree(deviceImage));
    checkCudaCall(hipFree(deviceHisto));

    hipEventSynchronize(s6);

    hipEventElapsedTime(&h2dTime, s1, s2);
    hipEventElapsedTime(&kernelTime, s3, s4);
    hipEventElapsedTime(&d2hTime, s5, s6);

    cout << "histogram (kernel): \t\t" << kernelTime / 1000 << " seconds."  << endl;
    cout << "histogram (memory): \t\t" << (h2dTime+d2hTime)/1000 << " seconds."  << endl;
/*
   hipEventDestroy(s1);
   hipEventDestroy(s2);
   hipEventDestroy(s3);
   hipEventDestroy(s4);
    hipEventDestroy(s5);
   hipEventDestroy(s6);
*/
}

void histogramSeq(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
  int i; 

  timer sequentialTime = timer("Sequential");
  
  for (i=0; i<hist_size; i++) histogram[i]=0;

  sequentialTime.start();
  for (i=0; i<img_size; i++) {
	histogram[image[i]]++;
  }
  sequentialTime.stop();
  
  cout << "histogram (sequential): \t\t" << sequentialTime << endl;

}

int main(int argc, char* argv[]) {
    long img_size = 655360;
    int hist_size = 256;
    
    if (argc > 1) img_size = atoi(argv[1]);
    if (img_size < 1024) {
	cout << "Error in parameter" << endl;
	exit(-1);
    }

    unsigned char *image = (unsigned char *)malloc(img_size * sizeof(unsigned char)); 
    unsigned int *histogramS = (unsigned int *)malloc(hist_size * sizeof(unsigned int));     
    unsigned int *histogram = (unsigned int *)malloc(hist_size * sizeof(unsigned int));

    // initialize the vectors.
    for(long i=0; i<img_size; i++) {
        image[i] = (unsigned char) (i % hist_size);
    }

    cout << "Compute the histogram of a gray image with " << img_size << " pixels." << endl;

    histogramSeq(image, img_size, histogramS, hist_size);
    histogramCuda(image, img_size, histogram, hist_size);
    
    // verify the resuls
    for(int i=0; i<hist_size; i++) {
	  if (histogram[i]!=histogramS[i]) {
            cout << "error in results! Bin " << i << " is "<< histogram[i] << ", but should be " << histogramS[i] << endl; 
            exit(1);
        }
    }
    cout << "results OK!" << endl;
     
    free(image);
    free(histogram);
    free(histogramS);         
    
    return 0;
}
