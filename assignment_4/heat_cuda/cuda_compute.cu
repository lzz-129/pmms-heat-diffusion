#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error \n");
        exit(1);
    }
}


__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
// insert operation here
    deviceResult[i] = deviceA[i]+deviceB[i];
}

extern "C" 
void cuda_do_compute() {
    int threadBlockSize = 512;
    int n=1024; 
    float a[1024], b[1024], result[1024];
    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        printf("Error in cudaMalloc! \n");
	return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        printf("Error in cudaMalloc! \n");
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        printf("Error in cudaMalloc! \n");
        return;
    }


    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
    hipDeviceSynchronize();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(b, deviceB, n * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

}
