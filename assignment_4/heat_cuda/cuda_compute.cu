#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "output.h"

#define BLOCK_SIZE 256
// #define GRID_SIZE 32
#define WARP_SIZE 32
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error \n");
        exit(1);
    }
}

__global__ void heatPaddingKernel(const size_t h, const size_t w, double *src) {
    
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= h) return;

    int curpos = tid * w;
    //one thread do one line
    src[curpos] = src[curpos + w - 2];
    src[curpos + w - 1] = src[curpos + 1];


}
__global__ void heatComputeKernel(const size_t h, const size_t w, double *src, double *dst, double *c) {
    
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;


    if(tid >= (h - 2) * (w - 2)) return;

    
    int curpos = tid + w + (int)(tid / (w-2)) * 2 + 1;
    int uppos = curpos - w;
    int lowpos = curpos + w;

    const double coef = c[curpos];
    const double restcoef = 1.0 - coef;
    const double c_cdir = 0.25 * M_SQRT2 / (M_SQRT2 + 1.0);
    const double c_cdiag = 0.25 / (M_SQRT2 + 1.0);

    dst[curpos] = coef * *(src + curpos) +
        /* direct neighbors */
        (*(src + curpos + 1) + *(src + curpos - 1) + 
        *(src + uppos) + *(src + lowpos)) * (restcoef * c_cdir) +
        /* diagonal neighbors */
        (*(src + uppos - 1) + *(src + uppos + 1) + 
        *(src + lowpos - 1) + *(src + lowpos + 1)) * (restcoef * c_cdiag);


}
__global__ void heatMaxdiffKernel(const size_t h, const size_t w, double *src, double *dst, double *output) {

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ double maxdiff[BLOCK_SIZE];
    

    if(tid >= (h - 2) * (w - 2)){
        maxdiff[threadIdx.x] = 0.0;
    }else{
        int curpos = tid + w + (int)(tid / (w-2)) * 2 + 1;
        maxdiff[threadIdx.x] = fabs(dst[curpos] - src[curpos]);
    }

    __syncthreads();


    for(unsigned int i = BLOCK_SIZE / 2; i > 0; i /= 2) {
        if(threadIdx.x < i){
            if(maxdiff[threadIdx.x + i] > maxdiff[threadIdx.x]) maxdiff[threadIdx.x] = maxdiff[threadIdx.x + i];
        }
        __syncthreads();
    }
    

    if(threadIdx.x == 0){
        output[blockIdx.x] = maxdiff[0];
    }
    
}

__global__ void heatFillReportKernel(const size_t h, const size_t w, double *src, double *dst, double *output) {

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ double sh_max[BLOCK_SIZE];
    __shared__ double sh_min[BLOCK_SIZE];
    __shared__ double sh_avg[BLOCK_SIZE];

    // printf("tid = %ld\n", tid);
    if(tid >= (h - 2) * (w - 2)){
        sh_max[threadIdx.x] = -INFINITY;
        sh_min[threadIdx.x] = INFINITY;
        sh_avg[threadIdx.x] = 0;
    }else{
        int curpos = tid + w + (int)(tid / (w-2)) * 2 + 1;
        double val = dst[curpos];
        sh_max[threadIdx.x] = val;
        sh_min[threadIdx.x] = val;
        sh_avg[threadIdx.x] = val;
    }

    __syncthreads();


    for(unsigned int i = BLOCK_SIZE / 2; i > 0; i /= 2) {
        if(threadIdx.x < i){
            if(sh_max[threadIdx.x + i] > sh_max[threadIdx.x]) sh_max[threadIdx.x] = sh_max[threadIdx.x + i];
            if(sh_min[threadIdx.x + i] < sh_min[threadIdx.x]) sh_min[threadIdx.x] = sh_min[threadIdx.x + i];
            sh_avg[threadIdx.x] += sh_avg[threadIdx.x + i];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0){
        output[blockIdx.x * 3] = sh_max[0];
        output[blockIdx.x * 3 + 1] = sh_min[0];
        output[blockIdx.x * 3 + 2] = sh_avg[0];
    }    

}

extern "C"
void cuda_do_report(const size_t h, const size_t w, double *src_dev, double *dst_dev, struct results *r)
{
    int block_size = BLOCK_SIZE;
    int grid_size = (int)ceil(((h - 2) * (w - 2))/(double)block_size);
    double *report_dev, report[grid_size][3];
    /* init globlal memory */
    checkCudaCall(hipMalloc((void **) &report_dev, 3 * grid_size * sizeof(double)));
    /* launch kernel */
    heatFillReportKernel<<<grid_size, block_size>>>(h, w, src_dev, dst_dev, report_dev);
    hipDeviceSynchronize();
    /* copy back result */
    checkCudaCall(hipMemcpy(report, report_dev, 3 * grid_size * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaCall(hipFree((void *)report_dev));
    double sum = 0;
    for(int i = 0; i < grid_size; i++){
        if(report[i][0] > r->tmax) r->tmax = report[i][0];
        if(report[i][1] < r->tmin) r->tmin = report[i][1];
        sum += report[i][2];
    }
    r->tavg = r->tavg  + (sum / ((h-2) * (w-2)));

}

extern "C" 
void cuda_do_init(size_t h, size_t w, double *src, double *dst, double *c, double **src_dev, double **dst_dev, double **c_dev){
    
    /* allocate device memory */
    checkCudaCall(hipMalloc((void **) src_dev, h * w * sizeof(double)));
    checkCudaCall(hipMalloc((void **) dst_dev, h * w * sizeof(double)));
    checkCudaCall(hipMalloc((void **) c_dev, h * w * sizeof(double)));


    /* copy memory to device */
    checkCudaCall(hipMemcpy(*src_dev, src, h * w * sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(*dst_dev, dst, h * w * sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(*c_dev, c, h * w * sizeof(double), hipMemcpyHostToDevice));

    hipDeviceSynchronize();
}
extern "C" 
double cuda_do_compute(size_t h, size_t w, double *src, double *dst, double *c){

    int block_size = BLOCK_SIZE;
    int grid_size = (int)ceil(((h - 2) * (w - 2))/(double)block_size);

    heatPaddingKernel<<<(int)ceil(h/(double)block_size), block_size>>>(h, w, src);
    hipDeviceSynchronize();

    heatComputeKernel<<<grid_size, block_size>>>(h, w, src, dst, c);
    hipDeviceSynchronize();

    double *maxdiff_dev, maxdiff[grid_size], diff = 0;
    /* init globlal memory */
    checkCudaCall(hipMalloc((void **) &maxdiff_dev, grid_size * sizeof(double)));
    /* launch kernel */
    heatMaxdiffKernel<<<grid_size, block_size>>>(h, w, src, dst, maxdiff_dev);
    hipDeviceSynchronize();
    /* copy back result */
    checkCudaCall(hipMemcpy(maxdiff, maxdiff_dev, grid_size * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaCall(hipFree((void *)maxdiff_dev));
    for(int i = 0; i < grid_size; i++){
        if(maxdiff[i] > diff) diff = maxdiff[i];
    }
    return diff;
}
extern "C" 
void cuda_do_deinit(double *src_dev, double *dst_dev, double *c_dev){

    checkCudaCall(hipFree((void *)src_dev));
    checkCudaCall(hipFree((void *)dst_dev));
    checkCudaCall(hipFree((void *)c_dev));

}